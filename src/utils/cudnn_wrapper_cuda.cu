#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// cudnn_wrapper .hpp .cpp - cuDNN support - wrapper classes, utility functions
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/cudnn_wrapper.hpp"
#include "lbann/utils/exception.hpp"

#include "El.hpp"

using namespace cudnn;
using namespace lbann;

namespace cudnn {

namespace {

__global__ void reduce_kernel(DataType *dst, const DataType *src,
                              El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  dst[offset] += src[offset];
}

}

void cudnn_manager::allreduce(const std::vector<DataType*>& gpu_data,
                              El::Int height,
                              El::Int width) {
  if (m_num_gpus < 2) {
    return;
  }

  const El::Int buf_len = 1 << 27;
  const El::Int work_len = buf_len * 2; // double buffering
  const El::Int work_len_bytes = work_len * sizeof(DataType);

  std::vector<DataType*> bufs[2];
  for(int i=0; i<m_num_gpus; ++i) {
    if (get_work_space_size(i) < work_len_bytes) {
      set_work_space_size(i, work_len_bytes); 
    }
    bufs[0].push_back(static_cast<DataType*>(get_work_space(i)));
    bufs[1].push_back(static_cast<DataType*>(get_work_space(i)) + buf_len);
  }  


  El::Int total_len = height * width;
  El::Int offset = 0;

  do {
    El::Int len = std::min(total_len - offset, buf_len);
    int sbuf_idx = 0;
    int dbuf_idx = 1;
    for (int j = 0; j < m_num_gpus - 1; ++j) {
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));
        int src_dev = i;
        int dst_dev = (i + 1) % m_num_gpus;              
        DataType *src_buf = j == 0 ? gpu_data[src_dev] + offset : bufs[sbuf_idx][src_dev];
        DataType *dst_buf = bufs[dbuf_idx][dst_dev];
        // copy to the next device in the ring
        FORCE_CHECK_CUDA(hipMemcpyPeerAsync(dst_buf, dst_dev, src_buf, src_dev,
                                             len * sizeof(DataType), get_stream(src_dev)));
      }
      synchronize();
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));        
        DataType *dst_buf = bufs[dbuf_idx][i];
        // TODO: use Thrust
        int tb_dim = 256;
        int grid_dim = len / tb_dim + (len % tb_dim ? 1 : 0);
        reduce_kernel<<<grid_dim, tb_dim>>>(gpu_data[i] + offset, dst_buf, len);
      }
      std::swap(sbuf_idx, dbuf_idx);
    }
    offset += len;
  } while (offset < total_len);
  
}

} // namespace cudnn
